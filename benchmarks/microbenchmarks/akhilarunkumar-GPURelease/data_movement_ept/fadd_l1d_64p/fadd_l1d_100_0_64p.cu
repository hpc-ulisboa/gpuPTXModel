#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#include <string>

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void shared_latency(float* D, int n, int div) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    double I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    if (thread_id < div) {
        __asm volatile (
                " .reg .f64 %r129;\n\t"
                " .reg .f64 %r113;\n\t"
                " .reg .f64 %r114;\n\t"
                " .reg .f64 %r115;\n\t"
                " .reg .f64 %r116;\n\t"
                " .reg .f64 %r117;\n\t"
                " .reg .f64 %r118;\n\t"
                " .reg .f64 %r119;\n\t"
                " .reg .f64 %r120;\n\t"
                " .reg .f64 %r121;\n\t"
                " .reg .f64 %r122;\n\t"
                " .reg .f64 %r123;\n\t"
                " .reg .f64 %r124;\n\t"
                " .reg .f64 %r125;\n\t"
                " .reg .f64 %r126;\n\t"
                " .reg .f64 %r127;\n\t"
                " .reg .f64 %r128;\n\t"
                "mov.f64 %r129, 4.4;\n\t"
                "mov.f64 %r113, %r129;\n\t"
                "mov.f64 %r114, 2.2;\n\t"
                "mov.f64 %r115, 3.3;\n\t"
                "mov.f64 %r116, 1.23;\n\t"
                "mov.f64 %r117, 2.42;\n\t"
                "mov.f64 %r118, 3.34;\n\t"
                "mov.f64 %r119, 5.62;\n\t"
                "mov.f64 %r120, 2.56;\n\t"
                "mov.f64 %r121, 1.56;\n\t"
                "mov.f64 %r122, 2.56;\n\t"
                "mov.f64 %r123, 5.56;\n\t"
                "mov.f64 %r124, 8.56;\n\t"
                "mov.f64 %r125, 3.56;\n\t"
                "mov.f64 %r126, 5.56;\n\t"
                "mov.f64 %r127, 6.56;\n\t"
                "mov.f64 %r128, 0.56;\n\t"

                );
        for (int k = 0; k < n; k++) {
            __asm volatile (
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    "add.rn.f64 %r113, %r129, %r113;\n\t"
                    "add.rn.f64 %r114, %r129, %r114;\n\t"
                    "add.rn.f64 %r115, %r129, %r115;\n\t"
                    "add.rn.f64 %r116, %r129, %r116;\n\t"
                    "add.rn.f64 %r117, %r129, %r117;\n\t"
                    "add.rn.f64 %r118, %r129, %r118;\n\t"
                    "add.rn.f64 %r119, %r129, %r119;\n\t"
                    "add.rn.f64 %r120, %r129, %r120;\n\t"
                    "add.rn.f64 %r121, %r129, %r121;\n\t"
                    "add.rn.f64 %r122, %r129, %r122;\n\t"
                    "add.rn.f64 %r123, %r129, %r123;\n\t"
                    "add.rn.f64 %r124, %r129, %r124;\n\t"
                    "add.rn.f64 %r125, %r129, %r125;\n\t"
                    "add.rn.f64 %r126, %r129, %r126;\n\t"
                    "add.rn.f64 %r127, %r129, %r127;\n\t"
                    "add.rn.f64 %r128, %r129, %r128;\n\t"
                    );
        }

//        double temp;
//        float output = 0.0;
//        asm("add.rn.f64 %0, r113, r114" : "=d"(temp));
//        asm("cvt.rn.f32.f64 %0, %1" : "=f"(output) : "d"(temp));
//        printf("%lf \n", output);
    }
    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = I1;

    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 6) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);
    int stride = atoi(argv[5]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(double));

//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipProfilerStart();

//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    shared_latency<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence);

    hipDeviceSynchronize();
    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipMemcpy(h_res, d_res, sizeof(double), hipMemcpyDeviceToHost);

    return 0;
}
