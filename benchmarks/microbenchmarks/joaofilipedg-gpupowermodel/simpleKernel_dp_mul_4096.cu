// Utilities and system includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define DATA_TYPE 1 // 0-SP, 1-INT, 2-DP
#define SIZE 60000000
#define TILE_DIM 1024

#define INNER_REPS 4096

template <class T> __global__ void simpleKernel(T *A, T *C1, T *C2, T *C3, T *C4)
{
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    T ra, rb, rc, rd;

    if (xIndex < SIZE) {
        ra=A[xIndex];
        rb=A[SIZE-xIndex];
        rc=A[xIndex];
        rd=A[SIZE-xIndex];

        // rb=A[xIndex];
        #pragma unroll 4096
        for (int i=0;i<INNER_REPS;i++) {
          ra=ra*rb;
          rb=rb*rc;
          rc=rc*rd;
          rd=rd*ra;
        }
        C1[xIndex]=ra;
        C2[xIndex]=rb;
        C3[xIndex]=rc;
        C4[xIndex]=rd;

    }
}


int main(int argc, char **argv) {
    int outer_reps, vector_size, tile_dim;
    vector_size = SIZE;
    tile_dim = TILE_DIM;

    if (argc>1){
        outer_reps = atoi(argv[1]);
    }else{
        outer_reps = 1;
    }

    // execution configuration parameters
    dim3 grid(vector_size/tile_dim, 1), threads(tile_dim, 1);

    // CUDA events
    hipEvent_t start, stop;

    size_t mem_size = static_cast<size_t>(sizeof(double) * vector_size);
    // allocate host memory
    double *h_iA = (double *) malloc(mem_size);
    double *h_oC1 = (double *) malloc(mem_size);
    double *h_oC2 = (double *) malloc(mem_size);
    double *h_oC3 = (double *) malloc(mem_size);
    double *h_oC4 = (double *) malloc(mem_size);
    // initalize host data
    for (int i = 0; i < vector_size; ++i)
    {
        h_iA[i] = (double) i+3;
        // h_iB[i] = (float) i+3;
    }
    // allocate device memory
    double *d_iA, *d_iB, *d_oC1, *d_oC2, *d_oC3, *d_oC4;

    hipMalloc((void **) &d_iA, mem_size);
    // cudaMalloc((void **) &d_iB, mem_size);
    hipMalloc((void **) &d_oC1, mem_size);
    hipMalloc((void **) &d_oC2, mem_size);
    hipMalloc((void **) &d_oC3, mem_size);
    hipMalloc((void **) &d_oC4, mem_size);

    // copy host data to device
    hipMemcpy(d_iA, h_iA, mem_size, hipMemcpyHostToDevice);
    // cudaMemcpy(d_iB, h_iB, mem_size, cudaMemcpyHostToDevice);

    // print out common data for all kernels
    printf("\nVector size: %d  TotalBlocks: %d blockSize: %d\n\n", vector_size, grid.x, threads.x);

    // initialize events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // take measurements for loop over kernel launches
    hipEventRecord(start, 0);

    for (int i=0; i < outer_reps; i++)
    {
        simpleKernel<double><<<grid, threads>>>(d_iA, d_oC1, d_oC2, d_oC3, d_oC4);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);

    // take measurements for loop inside kernel
    hipMemcpy(h_oC1, d_oC1, mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_oC2, d_oC2, mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_oC3, d_oC3, mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_oC4, d_oC4, mem_size, hipMemcpyDeviceToHost);

    printf("teste: %f\n", h_oC1[0]);

    // report effective bandwidths
    float kernelBandwidth = 2.0f * 1000.0f * mem_size/(1024*1024*1024)/(kernelTime/outer_reps);
    printf("simpleKernel, Throughput = %.4f GB/s, Time = %.5f ms, Size = %u fp32 elements, NumDevsUsed = %u, Workgroup = %u\n",
           kernelBandwidth,
           kernelTime/outer_reps,
           vector_size, 1, tile_dim * 1);

    free(h_iA);
    // free(h_iB);
    free(h_oC1);
    free(h_oC2);
    free(h_oC3);
    free(h_oC4);

    hipFree(d_iA);
    // cudaFree(d_iB);
    hipFree(d_oC1);
    hipFree(d_oC2);
    hipFree(d_oC3);
    hipFree(d_oC4);


    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();

    printf("Test passed\n");

    exit(EXIT_SUCCESS);
}
